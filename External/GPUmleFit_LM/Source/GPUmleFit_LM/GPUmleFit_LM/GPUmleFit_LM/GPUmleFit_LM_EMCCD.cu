#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "mex.h"
#include "hip/hip_runtime.h"
#include "definitions.h"
#include "MatInvLib.h"

#include "GPUsplineLib.cuh"
#include "GPUgaussLib.cuh"
#include "GPUmleFit_LM_EMCCD.h"


//#include "GPUsplineMLE.h"

__global__ void kernel_MLEFit_LM_EMCCD(const float *d_data,const float PSFSigma, const int sz, const int iterations, 
	float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){

		const int NV=NV_P;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;


		float model, data;
		float Div;
		float PSFy, PSFx;

		float newTheta[NV],oldTheta[NV];
		float newLambda = 1.0, oldLambda = 1.0;
		float newSign[NV] = {0}, oldSign[NV] = {0};
		float newUpdate[NV] = {0},oldUpdate[NV] = {0};
		float newClamp[NV]={1.0,1.0,100,20}, oldClamp[NV]={1.0,1.0,100,20};
		float newDudt[NV] ={0};

		float newErr = 1e13, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		float temp;
		int info;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);


		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues3D
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D(ii,jj,PSFSigma,newTheta,newDudt,&model);
			data=s_data[sz*jj+ii];

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else if (data==0)
				newErr = newErr + 2*model;

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}
		//addPeak

		//copyFitData

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//newStatus = CONVERGED;
				break;
			}
			else{
				if(newErr>1.5*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newSign[i]=oldSign[i];
						newClamp[i]=oldClamp[i];
						newTheta[i]=oldTheta[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;

					newLambda = 10*newLambda;
				}
				else if(newErr<oldErr){
					if (newLambda>1){
						newLambda = newLambda*0.8;
					}
					else if(newLambda<1){
						newLambda = 1;
					}
				}


				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*newLambda;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				info = kernel_cholesky(hessian,NV,L,U);
				if (info ==0){
					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);
					//copyFitData
					for (i=0;i<NV;i++){
						oldSign[i]=newSign[i];
						oldClamp[i]=newClamp[i];

						oldTheta[i]=newTheta[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;


					//updatePeakParameters
					for (ll=0;ll<NV;ll++){
						if (newSign[ll]!=0){
							if (newSign[ll]==1&&newUpdate[ll]<0){
								newClamp[ll]=newClamp[ll]*0.5;
							}
							else if (newSign[ll]==-1&&newUpdate[ll]>0){
								newClamp[ll] = newClamp[ll]*0.5;
							}
						}

						if (newUpdate[ll]>0){
							newSign[ll]=1;
						}
						else{
							newSign[ll]=-1;
						}

						newTheta[ll] = newTheta[ll]-newUpdate[ll]/(1+abs(newUpdate[ll]/newClamp[ll]));
					}

					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-2);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+2);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-2);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+2);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D(ii,jj,PSFSigma,newTheta,newDudt,&model);
						data=s_data[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else if (data==0)
							newErr = newErr + 2*model;

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					newLambda = 10*newLambda;
				}
			}
		}

		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			//need to check why don't use newTheta[4] instead of PSFSigma!!!
			kernel_DerivativeGauss2D(ii,jj,PSFSigma,newTheta,newDudt,&model);
			data=s_data[sz*jj+ii];

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//*********************************************************************************************************************************************
//Continue to optimize Hazen's workflow  change xc,xStart,yc,yStart calculation. Remove some redundency in kernel_computeDelta3D and kernal_fAt3D

__global__ void kernel_MLEFit_LM_Sigma_EMCCD(const float *d_data,const float PSFSigma, const int sz, const int iterations, 
	float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){

		const int NV=NV_PS;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;
		//int xstart, ystart, zstart, xi, yi;


		float model, data;
		float Div;
		float PSFy, PSFx;

		float newTheta[NV],oldTheta[NV];
		float newLambda = 1.0, oldLambda = 1.0;
		float newSign[NV] = {0}, oldSign[NV] = {0};
		float newUpdate[NV] = {0},oldUpdate[NV] = {0};
		float newClamp[NV]={1.0,1.0,100,20,0.5}, oldClamp[NV]={1.0,1.0,100,20,0.5};
		float newDudt[NV] ={0};

		float newErr = 1e13, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		float temp;
		int info;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
		newTheta[4]=PSFSigma;

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues3D
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
			data=s_data[sz*jj+ii];

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else if (data==0)
				newErr = newErr + 2*model;

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}
		//addPeak

		//copyFitData

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//newStatus = CONVERGED;
				break;
			}
			else{
				if(newErr>1.5*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newSign[i]=oldSign[i];
						newClamp[i]=oldClamp[i];
						newTheta[i]=oldTheta[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;

					newLambda = 10*newLambda;
				}
				else if(newErr<oldErr){
					if (newLambda>1){
						newLambda = newLambda*0.8;
					}
					else if(newLambda<1){
						newLambda = 1;
					}
				}


				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*newLambda;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				info = kernel_cholesky(hessian,NV,L,U);
				if (info ==0){
					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);
					//copyFitData
					for (i=0;i<NV;i++){
						oldSign[i]=newSign[i];
						oldClamp[i]=newClamp[i];

						oldTheta[i]=newTheta[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;


					//updatePeakParameters
					for (ll=0;ll<NV;ll++){
						if (newSign[ll]!=0){
							if (newSign[ll]==1&&newUpdate[ll]<0){
								newClamp[ll]=newClamp[ll]*0.5;
							}
							else if (newSign[ll]==-1&&newUpdate[ll]>0){
								newClamp[ll] = newClamp[ll]*0.5;
							}
						}

						if (newUpdate[ll]>0){
							newSign[ll]=1;
						}
						else{
							newSign[ll]=-1;
						}

						newTheta[ll] = newTheta[ll]-newUpdate[ll]/(1+abs(newUpdate[ll]/newClamp[ll]));
					}

					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-2);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+2);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-2);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+2);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					newTheta[4] = max(newTheta[4],0.0);
					newTheta[4] = min(newTheta[4],sz/2.0f);


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
						data=s_data[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else if (data==0)
							newErr = newErr + 2*model;

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					newLambda = 10*newLambda;
				}
			}
		}

		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			//need to check why don't use newTheta[4] instead of PSFSigma!!!
			//PSFx=kernel_IntGauss1D(ii, newTheta[0], PSFSigma);
			//PSFy=kernel_IntGauss1D(jj, newTheta[1], PSFSigma);

			//model=newTheta[3]+newTheta[2]*PSFx*PSFy;
			//data=s_data[sz*jj+ii];

			////calculating derivatives
			//kernel_DerivativeIntGauss1D(ii, newTheta[0], newTheta[4], newTheta[2], PSFy, &newDudt[0], NULL);
			//kernel_DerivativeIntGauss1D(jj, newTheta[1], newTheta[4], newTheta[2], PSFx, &newDudt[1], NULL);
			//kernel_DerivativeIntGauss2DSigma(ii, jj, newTheta[0], newTheta[1], newTheta[4], newTheta[2], PSFx, PSFy, &newDudt[4], NULL);
			//newDudt[2] = PSFx*PSFy;
			//newDudt[3] = 1.0f;
			kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
			data=s_data[sz*jj+ii];

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//*********************************************************************************************************************************************
//Continue to optimize Hazen's workflow  change xc,xStart,yc,yStart calculation. Remove some redundency in kernel_computeDelta3D and kernal_fAt3D

__global__ void kernel_MLEFit_LM_z_EMCCD(const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
	const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){

		const int NV=NV_PZ;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;
		//int xstart, ystart, zstart, xi, yi;


		float model, data;
		float Div;
		float PSFy, PSFx;

		float newTheta[NV],oldTheta[NV];
		float newLambda = 1.0, oldLambda = 1.0;
		float newSign[NV] = {0}, oldSign[NV] = {0};
		float newUpdate[NV] = {0},oldUpdate[NV] = {0};
		float newClamp[NV]={1.0,1.0,100,20,2}, oldClamp[NV]={1.0,1.0,100,20,2};
		float newDudt[NV] ={0};

		float newErr = 1e13, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		float temp;
		int info;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
		newTheta[4]=0;

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues3D
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			 kernel_DerivativeIntGauss2Dz(ii, jj, newTheta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, newDudt, NULL,&model);
			data=s_data[sz*jj+ii];

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else if (data==0)
				newErr = newErr + 2*model;

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}
		//addPeak

		//copyFitData

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//newStatus = CONVERGED;
				break;
			}
			else{
				if(newErr>1.5*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newSign[i]=oldSign[i];
						newClamp[i]=oldClamp[i];
						newTheta[i]=oldTheta[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;

					newLambda = 10*newLambda;
				}
				else if(newErr<oldErr){
					if (newLambda>1){
						newLambda = newLambda*0.8;
					}
					else if(newLambda<1){
						newLambda = 1;
					}
				}


				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*newLambda;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				info = kernel_cholesky(hessian,NV,L,U);
				if (info ==0){
					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);
					//copyFitData
					for (i=0;i<NV;i++){
						oldSign[i]=newSign[i];
						oldClamp[i]=newClamp[i];

						oldTheta[i]=newTheta[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;


					//updatePeakParameters
					for (ll=0;ll<NV;ll++){
						if (newSign[ll]!=0){
							if (newSign[ll]==1&&newUpdate[ll]<0){
								newClamp[ll]=newClamp[ll]*0.5;
							}
							else if (newSign[ll]==-1&&newUpdate[ll]>0){
								newClamp[ll] = newClamp[ll]*0.5;
							}
						}

						if (newUpdate[ll]>0){
							newSign[ll]=1;
						}
						else{
							newSign[ll]=-1;
						}

						newTheta[ll] = newTheta[ll]-newUpdate[ll]/(1+abs(newUpdate[ll]/newClamp[ll]));
					}

					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-2);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+2);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-2);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+2);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					//newTheta[4] = max(newTheta[4],0.0);
					//newTheta[4] = min(newTheta[4],sz/2.0f);


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
						data=s_data[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else if (data==0)
							newErr = newErr + 2*model;

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					newLambda = 10*newLambda;
				}
			}
		}

		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			//need to check why don't use newTheta[4] instead of PSFSigma!!!
		    kernel_DerivativeIntGauss2Dz(ii, jj, newTheta, PSFSigma_x,PSFSigma_y, Ax,Ay,Bx,By, gamma, d, &PSFx, &PSFy, newDudt, NULL,&model);
			data=s_data[sz*jj+ii];

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//*********************************************************************************************************************************************
//Continue to optimize Hazen's workflow  change xc,xStart,yc,yStart calculation. Remove some redundency in kernel_computeDelta3D and kernal_fAt3D

__global__ void kernel_MLEFit_LM_sigmaxy_EMCCD(const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){

		const int NV=NV_PS2;
		float M[NV*NV],Diag[NV], Minv[NV*NV];
		int tx = threadIdx.x;
		int bx = blockIdx.x;
		int BlockSize = blockDim.x;
		int ii, jj, kk, ll, l, m, i;
		//int xstart, ystart, zstart, xi, yi;


		float model, data;
		float Div;
		float PSFy, PSFx;

		float newTheta[NV],oldTheta[NV];
		float newLambda = 1.0, oldLambda = 1.0;
		float newSign[NV] = {0}, oldSign[NV] = {0};
		float newUpdate[NV] = {0},oldUpdate[NV] = {0};
		float newClamp[NV]={1.0,1.0,100,20,0.5,0.5}, oldClamp[NV]={1.0,1.0,100,20,0.5,0.5};
		float newDudt[NV] ={0};

		float newErr = 1e13, oldErr = 1e13;

		float jacobian[NV]={0};
		float hessian[NV*NV]={0};
		float t1,t2;

		float Nmax;
		float temp;
		int info;
		float L[NV*NV] = {0}, U[NV*NV] = {0};


		//Prevent read/write past end of array
		if ((bx*BlockSize+tx)>=Nfits) return;

		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

		//copy in data
		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);

		//initial values
		kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
		kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
		newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
		newTheta[4]=PSFSigma;
		newTheta[5]=PSFSigma;

		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

		//updateFitValues3D
		newErr = 0;
		memset(jacobian,0,NV*sizeof(float));
		memset(hessian,0,NV*NV*sizeof(float));
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			kernel_DerivativeGauss2D_sigmaxy( ii,  jj, PSFSigma, newTheta, newDudt, &model);
			data=s_data[sz*jj+ii];

			if (data>0)
				newErr = newErr + 2*((model-data)-data*log(model/data));
			else if (data==0)
				newErr = newErr + 2*model;

			t1 = 1-data/model;
			for (l=0;l<NV;l++){
				jacobian[l]+=t1*newDudt[l];
			}

			t2 = data/pow(model,2);
			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
				hessian[m*NV+l] = hessian[l*NV+m];
			}
		}
		//addPeak

		//copyFitData

		for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//newStatus = CONVERGED;
				break;
			}
			else{
				if(newErr>1.5*oldErr){
					//copy Fitdata

					for (i=0;i<NV;i++){
						newSign[i]=oldSign[i];
						newClamp[i]=oldClamp[i];
						newTheta[i]=oldTheta[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;

					newLambda = 10*newLambda;
				}
				else if(newErr<oldErr){
					if (newLambda>1){
						newLambda = newLambda*0.8;
					}
					else if(newLambda<1){
						newLambda = 1;
					}
				}


				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*newLambda;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				info = kernel_cholesky(hessian,NV,L,U);
				if (info ==0){
					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);
					//copyFitData
					for (i=0;i<NV;i++){
						oldSign[i]=newSign[i];
						oldClamp[i]=newClamp[i];

						oldTheta[i]=newTheta[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;


					//updatePeakParameters
					for (ll=0;ll<NV;ll++){
						if (newSign[ll]!=0){
							if (newSign[ll]==1&&newUpdate[ll]<0){
								newClamp[ll]=newClamp[ll]*0.5;
							}
							else if (newSign[ll]==-1&&newUpdate[ll]>0){
								newClamp[ll] = newClamp[ll]*0.5;
							}
						}

						if (newUpdate[ll]>0){
							newSign[ll]=1;
						}
						else{
							newSign[ll]=-1;
						}

						newTheta[ll] = newTheta[ll]-newUpdate[ll]/(1+abs(newUpdate[ll]/newClamp[ll]));
					}

					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-2);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+2);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-2);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+2);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					newTheta[4] = max(newTheta[4],PSFSigma/10.0f);
					newTheta[5] = max(newTheta[5],PSFSigma/10.0f);


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						//calculating derivatives
						kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
						data=s_data[sz*jj+ii];			

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else if (data==0)
							newErr = newErr + 2*model;

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					newLambda = 10*newLambda;
				}
			}
		}

		// Calculating the CRLB and LogLikelihood
		Div=0.0f;
		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
			//need to check why don't use newTheta[4] instead of PSFSigma!!!
			kernel_DerivativeGauss2D_sigmaxy( ii,  jj, PSFSigma, newTheta, newDudt, &model);
			data=s_data[sz*jj+ii];

			//Building the Fisher Information Matrix
			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
				M[ll*NV+kk]=M[kk*NV+ll];
			}

			//LogLikelyhood
			if (model>0)
				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
				else
					Div+=-model;
		}

		// Matrix inverse (CRLB=F^-1) and output assigments
		kernel_MatInvN(M, Minv, Diag, NV);
		//write to global arrays
		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
		d_LogLikelihood[BlockSize*bx+tx] = Div;

		return;
}

//******************************************************************************************************
//Continue to optimize Hazen's workflow  change xc,xStart,yc,yStart calculation. Remove some redundency in kernel_computeDelta3D and kernal_fAt3D

__global__ void kernel_splineMLEFit_z_EMCCD(const float *d_data,const float *d_coeff, const int spline_xsize, const int spline_ysize, const int spline_zsize, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits){
	
   const int NV=NV_PSP;
    float M[NV*NV],Diag[NV], Minv[NV*NV];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BlockSize = blockDim.x;
    int ii, jj, kk, ll, l, m, i;
	int xstart, ystart, zstart, xi, yi;

	const float *s_coeff;
	s_coeff = d_coeff;

    float model, data;
    float Div;
    //float dudt[NV_PS];
    float newTheta[NV],oldTheta[NV];
	float newLambda = 1.0, oldLambda = 1.0;
	float newSign[NV] = {0}, oldSign[NV] = {0};
	float newUpdate[NV] = {0},oldUpdate[NV] = {0};
	float newClamp[NV]={1.0,1.0,100,20,2}, oldClamp[NV]={1.0,1.0,100,20,2};
	float newDudt[NV] ={0};

	float newErr = 1e12, oldErr = 1e13;

	float off;
	float jacobian[NV]={0};
	float hessian[NV*NV]={0};
	float t1,t2;

	float Nmax;
	float xc,yc,zc;
	float temp;
	float delta_f[64]={0}, delta_dxf[64], delta_ddxf[64], delta_dyf[64], delta_ddyf[64], delta_dzf[64], delta_ddzf[64];
	int info;
	float L[NV*NV] = {0}, U[NV*NV] = {0};

    
    //Prevent read/write past end of array
    if ((bx*BlockSize+tx)>=Nfits) return;
    
    for (ii=0;ii<NV*NV;ii++)M[ii]=0;
    for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;

    //copy in data
      const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
	  //const float *s_varim = d_varim+(sz*sz*bx*BlockSize+sz*sz*tx);
	//const float *s_gainim = d_gainim+(sz*sz*bx*BlockSize+sz*sz*tx);
    
    //initial values
    kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
    kernel_GaussFMaxMin2D(sz, 1.5, s_data, &Nmax, &newTheta[3]);
    newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*1.5*1.5);
    newTheta[4]=float(spline_zsize)/2;

	for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];

	//updateFitValues3D
	xc = -2.0*((newTheta[0]-float(sz)/2)+0.5);
	yc = -2.0*((newTheta[1]-float(sz)/2)+0.5);

	off = (float(spline_xsize)+1.0-2*float(sz))/2;

	xstart = floor(xc);
	xc = xc-xstart;

	ystart = floor(yc);
	yc = yc-ystart;

	//zstart = floor(newTheta[4]);
	zstart = floor(newTheta[4]);
	zc = newTheta[4] -zstart;

	newErr = 0;
	memset(jacobian,0,NV*sizeof(float));
	memset(hessian,0,NV*NV*sizeof(float));
	kernel_computeDelta3D(xc, yc, zc, delta_f, delta_dxf, delta_dyf, delta_dzf);
	for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
		temp = kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart,spline_xsize,spline_ysize,spline_zsize,delta_f,s_coeff);
		model = newTheta[3]+newTheta[2]*temp;
		data=s_data[sz*jj+ii];
		//calculating derivatives

		newDudt[0] = -1*newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dxf,s_coeff);
		newDudt[1] = -1*newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dyf,s_coeff);
		newDudt[4] = newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dzf,s_coeff);
		newDudt[2] = temp;
		newDudt[3] = 1;

		if (data>0)
			newErr = newErr + 2*((model-data)-data*log(model/data));
		else if (data==0)
			newErr = newErr + 2*model;

		t1 = 1-data/model;
		for (l=0;l<NV;l++){
			jacobian[l]+=t1*newDudt[l];
		}

		t2 = data/pow(model,2);
		for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
			hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
			hessian[m*NV+l] = hessian[l*NV+m];
		}
	}
	//addPeak

	//copyFitData

	for (kk=0;kk<iterations;kk++) {//main iterative loop

			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
				//newStatus = CONVERGED;
				break;
			}
			else{
				if(newErr>1.5*oldErr){
					//copy Fitdata
					
					for (i=0;i<NV;i++){
						newSign[i]=oldSign[i];
						newClamp[i]=oldClamp[i];
						newTheta[i]=oldTheta[i];
					}
					newLambda = oldLambda;
					newErr = oldErr;

					newLambda = 10*newLambda;
				}
				else if(newErr<oldErr){
					if (newLambda>1){
						newLambda = newLambda*0.8;
					}
					else if(newLambda<1){
						newLambda = 1;
					}
				}
				

				for (i=0;i<NV;i++){
					hessian[i*NV+i]=hessian[i*NV+i]*newLambda;
				}
				memset(L,0,NV*sizeof(float));
				memset(U,0,NV*sizeof(float));
				info = kernel_cholesky(hessian,NV,L,U);
				if (info ==0){
					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);
                 //copyFitData
					for (i=0;i<NV;i++){
						oldSign[i]=newSign[i];
						oldClamp[i]=newClamp[i];

						oldTheta[i]=newTheta[i];
					}
					oldLambda = newLambda;
					oldErr=newErr;
					

					//updatePeakParameters
					for (ll=0;ll<NV;ll++){
						if (newSign[ll]!=0){
							if (newSign[ll]==1&&newUpdate[ll]<0){
								newClamp[ll]=newClamp[ll]*0.5;
							}
							else if (newSign[ll]==-1&&newUpdate[ll]>0){
								newClamp[ll] = newClamp[ll]*0.5;
							}
						}

						if (newUpdate[ll]>0){
							newSign[ll]=1;
						}
						else{
							newSign[ll]=-1;
						}

						newTheta[ll] = newTheta[ll]-newUpdate[ll]/(1+abs(newUpdate[ll]/newClamp[ll]));
					}

					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-2);
					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+2);
					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-2);
					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+2);
					newTheta[2] = max(newTheta[2],1.0);
					newTheta[3] = max(newTheta[3],0.01);
					newTheta[4] = max(newTheta[4],0.0);
					newTheta[4] = min(newTheta[4],float(spline_zsize));

					//updateFitValues3D
					xc = -2.0*((newTheta[0]-float(sz)/2)+0.5);
					yc = -2.0*((newTheta[1]-float(sz)/2)+0.5);

					off = (float(spline_xsize)+1.0-2*float(sz))/2;

					xstart = floor(xc);
					xc = xc-xstart;

					ystart = floor(yc);
					yc = yc-ystart;

					//zstart = floor(newTheta[4]);
					zstart = floor(newTheta[4]);
					zc = newTheta[4] -zstart;


					newErr = 0;
					memset(jacobian,0,NV*sizeof(float));
					memset(hessian,0,NV*NV*sizeof(float));
					kernel_computeDelta3D(xc, yc, zc, delta_f, delta_dxf, delta_dyf, delta_dzf);
					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
						temp = kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart,spline_xsize,spline_ysize,spline_zsize,delta_f,s_coeff);
						model = newTheta[3]+newTheta[2]*temp;
						data=s_data[sz*jj+ii];
						//calculating derivatives

						newDudt[0] = -1*newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dxf,s_coeff);
						newDudt[1] = -1*newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dyf,s_coeff);
						newDudt[4] = newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dzf,s_coeff);
						newDudt[2] = temp;
						newDudt[3] = 1;

						if (data>0)
							newErr = newErr + 2*((model-data)-data*log(model/data));
						else if (data==0)
							newErr = newErr + 2*model;

						t1 = 1-data/model;
						for (l=0;l<NV;l++){
							jacobian[l]+=t1*newDudt[l];
						}

						t2 = data/pow(model,2);
						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
							hessian[m*NV+l] = hessian[l*NV+m];
						}
					}
				}
				else
				{
					newLambda = 10*newLambda;
				}

					//copyFitdata	

			}


		
	}
    
    // Calculating the CRLB and LogLikelihood
	Div=0.0;

	xc = -2.0*((newTheta[0]-float(sz)/2)+0.5);
	yc = -2.0*((newTheta[1]-float(sz)/2)+0.5);

	off = (float(spline_xsize)+1.0-2*float(sz))/2;

	xstart = floor(xc);
	xc = xc-xstart;

	ystart = floor(yc);
	yc = yc-ystart;

	//zstart = floor(newTheta[4]);
	zstart = floor(newTheta[4]);
	zc = newTheta[4] -zstart;

	kernel_computeDelta3D(xc, yc, zc, delta_f, delta_dxf, delta_dyf, delta_dzf);

    for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
		temp = kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart,spline_xsize,spline_ysize,spline_zsize,delta_f,s_coeff);
		model = newTheta[3]+newTheta[2]*temp;
		data=s_data[sz*jj+ii];
		//calculating derivatives
		newDudt[0] = -1*newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dxf,s_coeff);
		newDudt[1] = -1*newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dyf,s_coeff);
		newDudt[4] = newTheta[2]*kernal_fAt3D(2*ii+xstart+off,2*jj+ystart+off,zstart, spline_xsize,spline_ysize,spline_zsize,delta_dzf,s_coeff);
		newDudt[2] = temp;
		newDudt[3] = 1;
        
        //Building the Fisher Information Matrix
        for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
            M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
            M[ll*NV+kk]=M[kk*NV+ll];
        }
        
        //LogLikelyhood
        if (model>0)
            if (data>0)Div+=data*log(model)-model-data*log(data)+data;
            else
                Div+=-model;
    }
    
    // Matrix inverse (CRLB=F^-1) and output assigments
    kernel_MatInvN(M, Minv, Diag, NV);
  
    
    //write to global arrays
    for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
   for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
   d_LogLikelihood[BlockSize*bx+tx] = Div;
	//d_LogLikelihood[BlockSize*bx+tx] = 1;
    
    
    return;
}



////*********************************************************************************************************************************************
////Continue to optimize Hazen's workflow  change xc,xStart,yc,yStart calculation. Remove some redundency in kernel_computeDelta3D and kernal_fAt3D
//
//__global__ void kernel_MLEFit_LM_EMCCD(const float *d_data,InitPara *para,float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int fittype){
//
//	
//		int ii, jj, kk, ll, l, m, i;
//		
//		//const float PSFSigma= para->PSFSigma;
//		//const int sz = para->sz;
//		//const int iterations = para->iterations;
//		//const int Nfits = para->Nfits;
//	    //const int NV = para->NV;
//		const int NV = 5;
//		//float *newClamp=(float*)malloc(NV*sizeof(float));
//		//float *oldClamp=(float*)malloc(NV*sizeof(float));
//		//for (ii=0;ii<NV;ii++){
//		//	newClamp[ii]=para->clamp[ii];
//		//	oldClamp[ii]=para->clamp[ii];
//		//}
//		float newClamp[NV]={1.0,1.0,100,20,0.5}, oldClamp[NV]={1.0,1.0,100,20,0.5};
//		float PSFSigma = 1.5;
//		int sz = 7;
//		int iterations = 20;
//		int Nfits = 10000;
//
//		//const float PSFSigma_x=para->PSFSigma_x;
//		//const float PSFSigma_y=para->PSFSigma_y;
//		//const float Ax=para->Ax;
//		//const float Ay=para->Ay;
//		//const float Bx=para->Bx;
//		//const float By=para->By;
//		//const float gamma=para->gamma;
//		//const float d=para->d;
//
//		//float *d_coeff=para->coeff;
//		//int spline_xsize=para->spline_xsize;
//		//int spline_ysize=para->spline_ysize;
//		//int spline_zsize=para->spline_ysize;
//		//float newTheta[NV],oldTheta[NV];
//		float *newTheta=(float*)malloc(NV*sizeof(float));
//		float *oldTheta=(float*)malloc(NV*sizeof(float));
//
//
//		float Nmax;
//		int xstart, ystart, zstart, xi, yi;
//		float xc,yc,zc;
//		float off;
//		float delta_f[64]={0}, delta_dxf[64], delta_ddxf[64], delta_dyf[64], delta_ddyf[64], delta_dzf[64], delta_ddzf[64];
//		
//
//		int tx = threadIdx.x;
//		int bx = blockIdx.x;
//		int BlockSize = blockDim.x;
//		//Prevent read/write past end of array
//		if ((bx*BlockSize+tx)>=Nfits) return;
//		//copy in data
//		const float *s_data = d_data+(sz*sz*bx*BlockSize+sz*sz*tx);
//
//
//		//***************************************************************************************************************
//		// Parameter initilaization
//		switch (fittype){
//		case 1:
//			//float newClamp[NV]={1.0,1.0,100,20}, oldClamp[NV]={1.0,1.0,100,20};
//
//			//kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
//			//kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
//			//newTheta[2]=max(0.0f, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
//
//		case 2:
//			//float newClamp[NV]={1.0,1.0,100,20,0.5}, oldClamp[NV]={1.0,1.0,100,20,0.5};
//
//			kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
//			kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
//			newTheta[2]=max(0.0f, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
//			newTheta[4]=PSFSigma;
//
//		case 3:
//			//float newClamp[NV]={1.0,1.0,100,20,1.0}, oldClamp[NV]={1.0,1.0,100,20,1.0};
//
//			//kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
//			//kernel_GaussFMaxMin2D(sz, PSFSigma_x, s_data, &Nmax, &newTheta[3]);
//			//newTheta[2]=max(0.0f, (Nmax-newTheta[3])*2*pi*PSFSigma_x*PSFSigma_y*sqrt(2.0f));
//			//newTheta[4]=0;
//
//		case 4:
//			//float newClamp[NV]={1.0,1.0,100,20,0.5,0.5}, oldClamp[NV]={1.0,1.0,100,20,0.5,0.5};
//
//			//kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
//			//kernel_GaussFMaxMin2D(sz, PSFSigma, s_data, &Nmax, &newTheta[3]);
//			//newTheta[2]=max(0.0f, (Nmax-newTheta[3])*2*pi*PSFSigma*PSFSigma);
//			//newTheta[4]=PSFSigma;
//			//newTheta[5]=PSFSigma;
//
//		case 5:
//			//float newClamp[NV]={1.0,1.0,100,20,2.0}, oldClamp[NV]={1.0,1.0,100,20,2.0};
//
//			//kernel_CenterofMass2D(sz, s_data, &newTheta[0], &newTheta[1]);
//			//kernel_GaussFMaxMin2D(sz, 1.5, s_data, &Nmax, &newTheta[3]);
//			//newTheta[2]=max(0.0, (Nmax-newTheta[3])*2*pi*1.5*1.5);
//			//newTheta[4]=float(spline_zsize)/2;
//
//		}
//
//		float M[NV*NV],Diag[NV], Minv[NV*NV];
//		float newSign[NV] = {0}, oldSign[NV] = {0};
//		float newUpdate[NV] = {0},oldUpdate[NV] = {0};	
//		float newDudt[NV] ={0};
//		//float *M=(float*)malloc(NV*NV*sizeof(float));
//		//float *Diag=(float*)malloc(NV*sizeof(float));
//		//float *Minv=(float*)malloc(NV*NV*sizeof(float));
//
//		//float *newSign=(float*)malloc(NV*sizeof(float));
//		//float *oldSign=(float*)malloc(NV*sizeof(float));
//		//float *newUpdate=(float*)malloc(NV*sizeof(float));
//		//float *oldUpdate=(float*)malloc(NV*sizeof(float));
//		//float *newDudt=(float*)malloc(NV*sizeof(float));
//		//float *jacobian=(float*)malloc(NV*sizeof(float));
//		//float *hessian=(float*)malloc(NV*NV*sizeof(float));
//		//float *L=(float*)malloc(NV*NV*sizeof(float));
//		//float *U=(float*)malloc(NV*NV*sizeof(float));
//
//		//memset(M,0,NV*NV*sizeof(float));
//		//memset(Diag,0,NV*sizeof(float));
//		//memset(Minv,0,NV*NV*sizeof(float));
//		//memset(newSign,0,NV*sizeof(float));
//		//memset(oldSign,0,NV*sizeof(float));
//		//memset(newUpdate,0,NV*sizeof(float));
//		//memset(oldUpdate,0,NV*sizeof(float));
//		//memset(newDudt,0,NV*sizeof(float));
//		//memset(jacobian,0,NV*sizeof(float));
//		//memset(hessian,0,NV*NV*sizeof(float));
//		//memset(L,0,NV*NV*sizeof(float));
//		//memset(U,0,NV*NV*sizeof(float));
//
//		float model, cf, df, data;
//		float Div;
//		float PSFy, PSFx;
//		
//		float newLambda = 1.0, oldLambda = 1.0;
//
//		float newErr = 1e12, oldErr = 1e13;
//
//		
//		float jacobian[NV]={0};
//		float hessian[NV*NV]={0};
//		float t1,t2;
//		
//		float temp;
//		
//		int info;
//		//float L[NV*NV] = {0}, U[NV*NV] = {0};
//			float *L= new float [NV*NV];
//		float *U=new float [NV*NV];
//				hipMemset (L,0,NV*NV*sizeof(float));
//		hipMemset (U,0,NV*NV*sizeof(float));
//
//		
//
//		for (ii=0;ii<NV*NV;ii++)M[ii]=0;
//		for (ii=0;ii<NV*NV;ii++)Minv[ii]=0;
//		for (ii=0;ii<NV;ii++)oldTheta[ii]=newTheta[ii];
//		
//
//		//updateFitValues3D
//		newErr = 0;
//		memset(jacobian,0,NV*sizeof(float));
//		memset(hessian,0,NV*NV*sizeof(float));
//		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {            
//			
//			kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
//			data=s_data[sz*jj+ii];
//
//			if (data>0)
//				newErr = newErr + 2*((model-data)-data*log(model/data));
//			else if (data==0)
//				newErr = newErr + 2*model;
//
//			t1 = 1-data/model;
//			for (l=0;l<NV;l++){
//				jacobian[l]+=t1*newDudt[l];
//			}
//
//			t2 = data/pow(model,2);
//			for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
//				hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
//				hessian[m*NV+l] = hessian[l*NV+m];
//			}
//		}
//		//addPeak
//
//		//copyFitData
//
//		for (kk=0;kk<iterations;kk++) {//main iterative loop
//
//			if(abs((newErr-oldErr)/newErr)<TOLERANCE){
//				//newStatus = CONVERGED;
//				break;
//			}
//			else{
//				if(newErr>1.5*oldErr){
//					//copy Fitdata
//
//					for (i=0;i<NV;i++){
//						newSign[i]=oldSign[i];
//						newClamp[i]=oldClamp[i];
//						newTheta[i]=oldTheta[i];
//					}
//					newLambda = oldLambda;
//					newErr = oldErr;
//
//					newLambda = 10*newLambda;
//				}
//				else if(newErr<oldErr){
//					if (newLambda>1){
//						newLambda = newLambda*0.8;
//					}
//					else if(newLambda<1){
//						newLambda = 1;
//					}
//				}
//
//
//				for (i=0;i<NV;i++){
//					hessian[i*NV+i]=hessian[i*NV+i]*newLambda;
//				}
//				memset(L,0,NV*sizeof(float));
//				memset(U,0,NV*sizeof(float));
//				info = kernel_cholesky(hessian,NV,L,U);
//				if (info ==0){
//					kernel_luEvaluate(L,U,jacobian,NV,newUpdate);
//					//copyFitData
//					for (i=0;i<NV;i++){
//						oldSign[i]=newSign[i];
//						oldClamp[i]=newClamp[i];
//
//						oldTheta[i]=newTheta[i];
//					}
//					oldLambda = newLambda;
//					oldErr=newErr;
//
//
//					//updatePeakParameters
//					for (ll=0;ll<NV;ll++){
//						if (newSign[ll]!=0){
//							if (newSign[ll]==1&&newUpdate[ll]<0){
//								newClamp[ll]=newClamp[ll]*0.5;
//							}
//							else if (newSign[ll]==-1&&newUpdate[ll]>0){
//								newClamp[ll] = newClamp[ll]*0.5;
//							}
//						}
//
//						if (newUpdate[ll]>0){
//							newSign[ll]=1;
//						}
//						else{
//							newSign[ll]=-1;
//						}
//
//						newTheta[ll] = newTheta[ll]-newUpdate[ll]/(1+abs(newUpdate[ll]/newClamp[ll]));
//					}
//
//					newTheta[0] = max(newTheta[0],(float(sz)-1)/2-2);
//					newTheta[0] = min(newTheta[0],(float(sz)-1)/2+2);
//					newTheta[1] = max(newTheta[1],(float(sz)-1)/2-2);
//					newTheta[1] = min(newTheta[1],(float(sz)-1)/2+2);
//					newTheta[2] = max(newTheta[2],1.0);
//					newTheta[3] = max(newTheta[3],0.01);
//					newTheta[4] = max(newTheta[4],0.0);
//					newTheta[4] = min(newTheta[4],sz/2.0f);
//
//
//					newErr = 0;
//					memset(jacobian,0,NV*sizeof(float));
//					memset(hessian,0,NV*NV*sizeof(float));
//					for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
//						//PSFx=kernel_IntGauss1D(ii, newTheta[0], newTheta[4]);
//						//PSFy=kernel_IntGauss1D(jj, newTheta[1], newTheta[4]);
//
//						//model=newTheta[3]+newTheta[2]*PSFx*PSFy;
//						//data=s_data[sz*jj+ii];
//
//						////calculating derivatives
//						//kernel_DerivativeIntGauss1D(ii, newTheta[0], newTheta[4], newTheta[2], PSFy, &newDudt[0],NULL);
//						//kernel_DerivativeIntGauss1D(jj, newTheta[1], newTheta[4], newTheta[2], PSFx, &newDudt[1],NULL);
//						//kernel_DerivativeIntGauss2DSigma(ii, jj, newTheta[0], newTheta[1], newTheta[4], newTheta[2], PSFx, PSFy, &newDudt[4],NULL);
//						//newDudt[2] = PSFx*PSFy;
//						//newDudt[3] = 1.0f;
//						/*(*jacf)(ii,jj,newTheta,newDudt,&model);*/
//						kernel_DerivativeGauss2D_sigma(ii,jj,newTheta,newDudt,&model);
//						//(*jacf)(ii,jj,newTheta,newDudt,&model);
//						data=s_data[sz*jj+ii];
//
//						if (data>0)
//							newErr = newErr + 2*((model-data)-data*log(model/data));
//						else if (data==0)
//							newErr = newErr + 2*model;
//
//						t1 = 1-data/model;
//						for (l=0;l<NV;l++){
//							jacobian[l]+=t1*newDudt[l];
//						}
//
//						t2 = data/pow(model,2);
//						for (l=0;l<NV;l++) for(m=l;m<NV;m++) {
//							hessian[l*NV+m] +=t2*newDudt[l]*newDudt[m];
//							hessian[m*NV+l] = hessian[l*NV+m];
//						}
//					}
//				}
//				else
//				{
//					newLambda = 10*newLambda;
//				}
//
//				//copyFitdata
//			}
//		}
//
//		// Calculating the CRLB and LogLikelihood
//		Div=0.0f;
//		for (ii=0;ii<sz;ii++) for(jj=0;jj<sz;jj++) {
//			//need to check why don't use newTheta[4] instead of PSFSigma!!!
//			PSFx=kernel_IntGauss1D(ii, newTheta[0], PSFSigma);
//			PSFy=kernel_IntGauss1D(jj, newTheta[1], PSFSigma);
//
//			model=newTheta[3]+newTheta[2]*PSFx*PSFy;
//			data=s_data[sz*jj+ii];
//
//			//calculating derivatives
//			kernel_DerivativeIntGauss1D(ii, newTheta[0], newTheta[4], newTheta[2], PSFy, &newDudt[0], NULL);
//			kernel_DerivativeIntGauss1D(jj, newTheta[1], newTheta[4], newTheta[2], PSFx, &newDudt[1], NULL);
//			kernel_DerivativeIntGauss2DSigma(ii, jj, newTheta[0], newTheta[1], newTheta[4], newTheta[2], PSFx, PSFy, &newDudt[4], NULL);
//			newDudt[2] = PSFx*PSFy;
//			newDudt[3] = 1.0f;
//
//			//Building the Fisher Information Matrix
//			for (kk=0;kk<NV;kk++)for (ll=kk;ll<NV;ll++){
//				M[kk*NV+ll]+= newDudt[ll]*newDudt[kk]/model;
//				M[ll*NV+kk]=M[kk*NV+ll];
//			}
//
//			//LogLikelyhood
//			if (model>0)
//				if (data>0)Div+=data*log(model)-model-data*log(data)+data;
//				else
//					Div+=-model;
//		}
//
//		// Matrix inverse (CRLB=F^-1) and output assigments
//		kernel_MatInvN(M, Minv, Diag, NV);
//		//write to global arrays
//		for (kk=0;kk<NV;kk++) d_Parameters[Nfits*kk+BlockSize*bx+tx]=newTheta[kk];
//		for (kk=0;kk<NV;kk++) d_CRLBs[Nfits*kk+BlockSize*bx+tx]=Diag[kk];
//		d_LogLikelihood[BlockSize*bx+tx] = Div;
//
//		return;
//}
//
