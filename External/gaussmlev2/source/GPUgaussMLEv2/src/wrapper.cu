/*!
 * \file wrapper.cu
 * \author Keith Lidke
 * \date January 10, 2010
 * \brief Wrap the Cuda kernel calls as standard external C functions.  This allows the kernels to be
 * called without doing anything special in the C code and simplifies building the code.
 */

#include "GPUgaussMLEv2.h"

//*******************************************************************************************
extern "C" void kernel_MLEFit_wrapper(dim3 dimGrid, dim3 dimBlock, float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */

	kernel_MLEFit<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern "C" void kernel_MLEFit_sigma_wrapper(dim3 dimGrid, dim3 dimBlock, float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_sigma<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern "C" void kernel_MLEFit_z_wrapper(dim3 dimGrid, dim3 dimBlock, float *d_data, float PSFSigma_x, float Ax, float Ay, float Bx, 
		float By, float gamma, float d, float PSFSigma_y, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma_x the sigma value to use for the point spread function on the x axis
 *  \param Ax ???
 *  \param Ay ???
 *  \param Bx ???
 *  \param By ???
 *  \param gamma ???
 *  \param d ???
 *  \param PSFSigma_y the sigma value to use for the point spread function on the y axis
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_z<<<dimGrid, dimBlock>>>(d_data, PSFSigma_x, Ax, Ay, Bx, By, gamma, d, PSFSigma_y, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern "C" void kernel_MLEFit_sigmaxy_wrapper(dim3 dimGrid, dim3 dimBlock, float *d_data, float PSFSigma, int sz, int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,int Nfits) 
{
	/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_sigmaxy<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}