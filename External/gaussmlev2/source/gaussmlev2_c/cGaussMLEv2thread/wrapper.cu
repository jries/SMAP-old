/*!
 * \file wrapper.cu
 * \author Keith Lidke
 * \date January 10, 2010
 * \brief Wrap the Cuda kernel calls as standard external C functions.  This allows the kernels to be
 * called without doing anything special in the C code and simplifies building the code.
 */

#include "GPUgaussMLEv2.h"

//*******************************************************************************************
extern void kernel_MLEFit_noshared_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */

	kernel_MLEFit_noshared<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern void kernel_MLEFit_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */

	kernel_MLEFit<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern void kernel_MLEFit_sigma_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_sigma<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern void kernel_MLEFit_z_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma_x, const float Ax, const float Ay, const float Bx, 
		const float By, const float gamma, const float d, const float PSFSigma_y, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma_x the sigma value to use for the point spread function on the x axis
 *  \param Ax ???
 *  \param Ay ???
 *  \param Bx ???
 *  \param By ???
 *  \param gamma ???
 *  \param d ???
 *  \param PSFSigma_y the sigma value to use for the point spread function on the y axis
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_z<<<dimGrid, dimBlock>>>(d_data, PSFSigma_x, Ax, Ay, Bx, By, gamma, d, PSFSigma_y, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}

//*******************************************************************************************
extern void kernel_MLEFit_sigmaxy_wrapper(dim3 dimGrid, dim3 dimBlock, const float *d_data, const float PSFSigma, const int sz, const int iterations, 
        float *d_Parameters, float *d_CRLBs, float *d_LogLikelihood,const int Nfits) 
{
	/*!
 *  \brief Basic maximum likelihood estimator fit based kernel
 *  \param dimGrid number of blocks 
 *  \param dimBlock number of threads per block
 *  \param d_data an array of subregions to be processed copied into video memory
 *  \param PSFSigma the sigma value to use for the point spread function
 *  \param sz nxn size of the subregion
 *  \param iterations maximum allowed iterations before aborting fitting
 *  \param d_Parameters pointer to result array of fitted parameters, x, y coords, etc.
 *  \param d_CRLBs pointer to result array of Cramer-Rao lower bound estimates 
 *  \param d_LogLikelihood pointer to result array of loglikelihood estimates of fitting
 *  \param Nfits number of subregions to fit
 */
	kernel_MLEFit_sigmaxy<<<dimGrid, dimBlock>>>(d_data, PSFSigma, sz, iterations, d_Parameters, d_CRLBs, d_LogLikelihood, Nfits);
}